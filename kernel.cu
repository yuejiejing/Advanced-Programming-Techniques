#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <complex.h>
#include <input_image.h>
#include <hip/device_functions.h>

// #include <ctime>
// #include <chrono>

using namespace std;

#define Pi 3.14159265358979f


static __device__ __host__ inline void complexMul(const float &ar, const float &ai, const float &br,
												  const float &bi, float &cr, float &ci) {
	cr = ar * br - ai * bi;
	ci = ar * bi + ai * br;
}


__global__ void dft1D(float* real, float* imag, const int w) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int x = threadIdx.x;

	extern __shared__ float cache[];

	cache[x] = real[idx];
	cache[x + w] = imag[idx];

	__syncthreads();

	float Wr = 0;
	float Wi = 0;
	float Hr = 0;
	float Hi = 0;
	float cul_Hr = 0;
	float cul_Hi = 0;

	for (int k = 0; k < w; ++k) {
		Wr = cosf(2 * Pi * x * (float)k / (float)w);
		Wi = -sinf(2 * Pi * x * (float)k / (float)w);

		complexMul(Wr, Wi, cache[k], cache[k + w], Hr, Hi);
		cul_Hr += Hr;
		cul_Hi += Hi;
	}

	real[idx] = cul_Hr;
	imag[idx] = cul_Hi;
}


__global__ void transpose(float* r, float* i, int w) {
	int x = threadIdx.x;
	int y = blockIdx.x;

	if (y > x) {
		float temp_i = i[y * w + x];
		float temp_r = r[y * w + x];
		r[y * w + x] = r[x * w + y];
		r[x * w + y] = temp_r;
		i[y * w + x] = i[x * w + y];
		i[x * w + y] = temp_i;
	}
}



int main(int argc, char** argv) {

	// auto start = chrono::system_clock::now();
	
	// Argument Parsing
	if (argc == 1) {
		cout << "\nNo argument was passed.\n";
		exit(1);
	}
	else if (argc != 4) {
		cout << "\nThe number of argument is incorrect.\n";
		exit(1);
	}

	// Parse parameter
	char * in_name(argv[2]);
	char * out_name(argv[3]);

	bool is_forward;

	if (!strcmp(argv[1], "forward")) is_forward = true;
	else if (!strcmp(argv[1], "reverse")) is_forward = false;
	else {
		cout << "Parameter incorrect!" << endl;
		exit(1);
	}

	if (is_forward == false) {
		cout << "Reverse mode is not supported in this project due to incorrect image read function." << endl;
		exit(0);
	}

	// import virtual image
	InputImage img(in_name);
	int img_width = img.get_width();
	int img_height = img.get_height();
	if (img_width != img_height) {
		cout << "The input image is not square!" << endl;
		exit(1);
	}
	if (log2((float)img_width) != floor(log2((float)img_width))) {
		cout << "The input resolution must equal to 2 ^ n!" << endl;
		exit(1);
	}
	cout << "image read: success" << endl;

	int size_1d = img_width * img_width;

	float *h_real = reinterpret_cast<float *>(malloc(sizeof(float) * size_1d));
	float *h_imag = reinterpret_cast<float *>(malloc(sizeof(float) * size_1d));

	// initialize with image data
	Complex * img_data = img.get_image_data();
	for (int i = 0; i < size_1d; ++i) {
		h_real[i] = img_data[i].real;
		h_imag[i] = img_data[i].imag;
	}

	// Allocate device memory
	float *d_real, *d_imag;
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_real), size_1d * sizeof(float)));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_imag), size_1d * sizeof(float)));

	// copy host memory to device
	checkCudaErrors(hipMemcpy(d_real, h_real, size_1d * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_imag, h_imag, size_1d * sizeof(float), hipMemcpyHostToDevice));

	// int block_size = img_width;
	// int block_num = img_width;


	dft1D <<< img_width, img_width, img_width * sizeof(float) * 2 >>> (d_real, d_imag, img_width);
	transpose <<< img_width, img_width >>> (d_real, d_imag, img_width);
	dft1D <<< img_width, img_width, img_width * sizeof(float) * 2 >>> (d_real, d_imag, img_width);
	transpose <<< img_width, img_width >>> (d_real, d_imag, img_width);

	checkCudaErrors(hipMemcpy(h_real, d_real, size_1d * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_imag, d_imag, size_1d * sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < size_1d; ++i) {
		img_data[i].real = h_real[i];
		img_data[i].imag = h_imag[i];
	}

	img.save_image_data(out_name, img_data, img_width, img_height);

	cout << "Compute Finished" << endl;
	free(h_real);
	free(h_imag);
	checkCudaErrors(hipFree(d_real));
	checkCudaErrors(hipFree(d_imag));

	// auto end = chrono::system_clock::now();
	// chrono::duration<double> elapsed_time = end-start;
	// cout << "elapsed_time = " << elapsed_time.count() << endl;

    return 0;
}
